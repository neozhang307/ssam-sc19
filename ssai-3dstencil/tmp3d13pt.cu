#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cassert>
#include <cstdio>
#include "common.hpp"


////////////////////////////////////
#if 1
#define    _F3d13PT_022         ((T)0.100000)
#define    _F3d13PT_122         ((T)0.200000)
#define    _F3d13PT_202         ((T)0.700000)
#define    _F3d13PT_212         ((T)0.600000)
#define    _F3d13PT_220         ((T)1.000000)
#define    _F3d13PT_221         ((T)1.100000)
#define    _F3d13PT_222         ((T)0.300000)
#define    _F3d13PT_223         ((T)1.200000)
#define    _F3d13PT_224         ((T)1.300000)
#define    _F3d13PT_232         ((T)0.800000)
#define    _F3d13PT_242         ((T)0.900000)
#define    _F3d13PT_322         ((T)0.400000)
#define    _F3d13PT_422         ((T)0.500000)
#else
#pragma message("using debug parameters")
#define    _F3d13PT_022         0.00000
#define    _F3d13PT_122         0.00000
#define    _F3d13PT_202         0.00000
#define    _F3d13PT_212         0.00000
#define    _F3d13PT_220         0.00000
#define    _F3d13PT_221         0.00000
#define    _F3d13PT_222         1.00000
#define    _F3d13PT_223         0.00000
#define    _F3d13PT_224         0.00000
#define    _F3d13PT_232         0.00000
#define    _F3d13PT_242         0.00000
#define    _F3d13PT_322         0.00000
#define    _F3d13PT_422         0.00000
#endif
////////////////////////////////////
#define _FILTER_SIZE  5
const int FILTER_WIDTH = _FILTER_SIZE;
const int FILTER_HEIGHT = _FILTER_SIZE;
const int FILTER_DEPTH = _FILTER_SIZE;
////////////////////////////////////

//typedef double REAL;
static const int WARP_SIZE = 32;

#define max(x,y)  ((x) > (y)? (x) : (y))
#define min(x,y)  ((x) < (y)? (x) : (y))
#define ceil(a,b) ((a) % (b) == 0 ? (a) / (b) : ((a) / (b)) + 1)

__device__ __host__ __forceinline__ unsigned int UpDivide(unsigned int x, unsigned int y) { assert(y != 0); return (x + y - 1) / y; }
__device__ __host__ __forceinline__ unsigned int UpRound(unsigned int x, unsigned int y) { return UpDivide(x, y)*y; }
#if (CUDART_VERSION >= 9000)
#pragma message("CUDART_VERSION >= 9000")
#define __my_shfl_up(var, delta) __shfl_up_sync(0xFFFFFFFF, var, delta)
#define __my_shfl_down(var, delta) __shfl_down_sync(0xFFFFFFFF, var, delta)
#else
#pragma message("CUDART_VERSION < 9000")
#define __my_shfl_up(var, delta) __shfl_up(var, delta)
#define __my_shfl_down(var, delta) __shfl_down(var, delta)
#endif

#define MAD(__x, __y, __z) ((__x)*(__y)+(__z))

static void check_error(const char* message) {
	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		printf("CUDA error : %s, %s\n", message, hipGetErrorString(error));
		exit(-1);
	}
}

template<typename T, int FILTER_SIZE, int PROCESS_DATA_COUNT_Y, int PROCESS_DATA_COUNT_Z, int WARP_COUNT, int ITERATIVE_COUNT>
__global__ void kernel3d_3d13pt(const T * __restrict__ src, T* dst,
	int nx, int ny, int nz, int nxy) {
		//assert(ITERATIVE_COUNT == 1);
		const int FILTER_WIDTH = FILTER_SIZE;
		const int FILTER_HEIGHT = FILTER_SIZE;
		const int FILTER_DEPTH = FILTER_SIZE;
		const int HALF_FILTER_WIDTH = FILTER_WIDTH / 2;
		const int HALF_FILTER_HEIGHT = FILTER_HEIGHT / 2;
		const int HALF_FILTER_DEPTH = FILTER_DEPTH / 2;
#ifdef _DEBUG
		const int laneId = threadIdx.x;
		const int warpId = threadIdx.y;
#else
#define laneId  (threadIdx.x)
#define warpId  (threadIdx.y)
#endif
		const int WARP_PROCESS_DATA_COUNT_X = WARP_SIZE - (FILTER_WIDTH - 1) * ITERATIVE_COUNT;
		const int DATA_CACHE_SIZE = PROCESS_DATA_COUNT_Y + (FILTER_HEIGHT - 1) * ITERATIVE_COUNT;
		assert(WARP_COUNT == PROCESS_DATA_COUNT_Z + (FILTER_SIZE - 1) * ITERATIVE_COUNT);

		//__shared__ T sMem[WARP_COUNT][DATA_CACHE_SIZE][WARP_SIZE];
		//T* p = sMem[0][0];

		__shared__ T sMem[WARP_COUNT][DATA_CACHE_SIZE- HALF_FILTER_HEIGHT*2][WARP_SIZE- HALF_FILTER_WIDTH*2];

		T data[DATA_CACHE_SIZE];

#ifdef _DEBUG
		const int width = nx;
		const int height = ny;
		const int depth = nz;
#else
#define width nx
#define height ny
#define depth  nz
#endif
		const int process_count_x = WARP_PROCESS_DATA_COUNT_X*blockIdx.x;
		const int process_count_y = PROCESS_DATA_COUNT_Y*blockIdx.y;
		const int process_count_z = PROCESS_DATA_COUNT_Z*blockIdx.z;

		const int tidx = process_count_x + laneId - HALF_FILTER_WIDTH * ITERATIVE_COUNT;
		const int tidy = process_count_y - HALF_FILTER_HEIGHT * ITERATIVE_COUNT;
		const int tidz = process_count_z + warpId - HALF_FILTER_DEPTH * ITERATIVE_COUNT;

		int index = nxy*tidz + nx*tidy + tidx;
		{
			if (tidx < 0)        index -= tidx;
			else if (tidx >= nx) index -= tidx - nx + 1;
			if (tidz < 0)        index -= tidz*nxy;
			else if (tidz >= nz) index -= (tidz - nz + 1)*nxy;
			if (tidy < 0)        index -= tidy*nx;
			else if (tidy >= ny) index -= (tidy - ny + 1)*nx;

#pragma unroll
			for (int s = 0; s < DATA_CACHE_SIZE; s++) {
				int _tidy = tidy + s;
				data[s] = src[index];
				if (_tidy >= 0 && _tidy < ny - 1) index += nx;
			}
		}

		//iterative compute
		#pragma unroll
		for (int ite = 0; ite < ITERATIVE_COUNT; ite++) {
			if (laneId >= HALF_FILTER_WIDTH && laneId < WARP_SIZE - HALF_FILTER_WIDTH) {
				#pragma unroll
				for (int i = HALF_FILTER_HEIGHT; i < DATA_CACHE_SIZE - HALF_FILTER_HEIGHT - HALF_FILTER_HEIGHT*2*ite; i++) {
					sMem[warpId][i- HALF_FILTER_HEIGHT][laneId- HALF_FILTER_WIDTH] = data[i];
				}
			}
			__syncthreads();

			if (warpId < HALF_FILTER_DEPTH*(1 + ite) || warpId >= WARP_COUNT - HALF_FILTER_DEPTH*(1 + ite)) {
				return;
			}
			const int CURRENT_PROCESS_DATA = DATA_CACHE_SIZE - (FILTER_HEIGHT - 1)*(ite + 1);

			#pragma unroll
			for (int i = 0; i < CURRENT_PROCESS_DATA; i++) {
				T sum = 0;
				{
					//m = 0
					sum = MAD(data[i + 2], _F3d13PT_220, sum);
				}
				{
					//m = 1
					sum = __my_shfl_up(sum, 1);
					sum = MAD(data[i + 2], _F3d13PT_221, sum);
				}
				{
					//m = 4
					sum = __my_shfl_up(sum, 3);
					sum = MAD(data[i + 2], _F3d13PT_224, sum);
				}
				{
					//m = 3
					sum = __my_shfl_down(sum, 1);
					sum = MAD(data[i + 2], _F3d13PT_223, sum);
				}
				{
					//m = 2
					sum = __my_shfl_down(sum, 1);
					sum = MAD(data[i + 0], _F3d13PT_202, sum);
					sum = MAD(data[i + 1], _F3d13PT_212, sum);
					sum = MAD(data[i + 2], _F3d13PT_222, sum);
					sum = MAD(data[i + 3], _F3d13PT_232, sum);
					sum = MAD(data[i + 4], _F3d13PT_242, sum);
				}
				if (laneId >= HALF_FILTER_WIDTH*(ite+1) && laneId < WARP_SIZE - HALF_FILTER_WIDTH*(ite + 1)){
					sum += sMem[warpId - 2][i + HALF_FILTER_HEIGHT- HALF_FILTER_HEIGHT][laneId - HALF_FILTER_WIDTH] * _F3d13PT_022;
					sum += sMem[warpId - 1][i + HALF_FILTER_HEIGHT - HALF_FILTER_HEIGHT][laneId - HALF_FILTER_WIDTH] * _F3d13PT_122;
					sum += sMem[warpId + 1][i + HALF_FILTER_HEIGHT - HALF_FILTER_HEIGHT][laneId - HALF_FILTER_WIDTH] * _F3d13PT_322;
					sum += sMem[warpId + 2][i + HALF_FILTER_HEIGHT - HALF_FILTER_HEIGHT][laneId - HALF_FILTER_WIDTH] * _F3d13PT_422;
					data[i] = sum;
				}
			}
			__syncthreads();
		}

		//if (warpId < HALF_FILTER_DEPTH * ITERATIVE_COUNT || warpId >= WARP_COUNT - HALF_FILTER_DEPTH * ITERATIVE_COUNT) {
		//	assert(0);
		//}
		if (laneId < HALF_FILTER_WIDTH * ITERATIVE_COUNT || laneId >= WARP_SIZE - HALF_FILTER_WIDTH * ITERATIVE_COUNT)
			return;

		//save to gmem
		{
#define _tidx tidx
			int _tidy = tidy + HALF_FILTER_HEIGHT * ITERATIVE_COUNT;
#define _tidz tidz
			//int _tidx = tidx - (FILTER_WIDTH - 1) / 2*2;
			index = _tidz*nxy + nx*_tidy + _tidx;
#pragma unroll
			for (int i = 0; i < PROCESS_DATA_COUNT_Y; i++) {
				{
					assert(_tidx >= 0);
					assert(_tidy >= 0);
					assert(_tidz >= 0);
					if (_tidx < nx && _tidy < ny && _tidz < nz) {
						//dst[index] = sMem[warpId][i][laneId];
						dst[index] = data[i];
					}
				}
				_tidy++;
				index += nx;
			}
		}
#ifdef _DEBUG
#else
#undef laneId
#undef warpId
#undef width
#undef height
#undef depth
#endif
}



template<typename T, int ITERATIVE_COUNT, int BLOCK_SIZE, int CACHE_DATA_COUNT>
static void host_code(T *h_in, T *h_out, int N) {
	T *in;
	hipMalloc(&in, sizeof(T)*N*N*N);
	check_error("Failed to allocate device memory for in\n");
	hipMemcpy(in, h_in, sizeof(T)*N*N*N, hipMemcpyHostToDevice);
	T *out;
	hipMalloc(&out, sizeof(T)*N*N*N);
	check_error("Failed to allocate device memory for out\n");

	{
		//V100, It 3, 14 24
#if 0
		const int CACHE_DATA_COUNT = 16;
		const int WARP_COUNT = 24;
#else
		//const int CACHE_DATA_COUNT = 12;
		//const int WARP_COUNT = 16;
		const int WARP_COUNT = BLOCK_SIZE/WARP_SIZE;
#endif
		const int PROCESS_DATA_COUNT = CACHE_DATA_COUNT - (FILTER_WIDTH - 1)*ITERATIVE_COUNT;
		const int BLOCK_PROCESS_DATA_COUNT_X = WARP_SIZE - (FILTER_WIDTH - 1)*ITERATIVE_COUNT;
		const int BLOCK_PROCESS_DATA_COUNT_Y = PROCESS_DATA_COUNT;
		const int BLOCK_PROCESS_DATA_COUNT_Z = WARP_COUNT - (FILTER_DEPTH - 1)*ITERATIVE_COUNT;

		{
			assert(WARP_COUNT >= FILTER_DEPTH);

			int flag = 0;
			const int nx_ = N;
			const int ny_ = N;
			const int nz_ = N;
			size_t s = sizeof(T) * nx_ * ny_ * nz_;
			dim3 block_size(WARP_SIZE, WARP_COUNT);
			dim3 grid_size(UpDivide(nx_, BLOCK_PROCESS_DATA_COUNT_X),
				UpDivide(ny_, BLOCK_PROCESS_DATA_COUNT_Y),
				UpDivide(nz_, BLOCK_PROCESS_DATA_COUNT_Z)
			);
			float ftime = 0;
			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);

			hipEventRecord(start, 0);
			kernel3d_3d13pt<T, _FILTER_SIZE, BLOCK_PROCESS_DATA_COUNT_Y, BLOCK_PROCESS_DATA_COUNT_Z, WARP_COUNT, ITERATIVE_COUNT> << <grid_size, block_size >> >
				(in, out, nx_, ny_, nz_, nx_*ny_);

			hipDeviceSynchronize();
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			hipEventElapsedTime(&ftime, start, stop);

			hipMemcpy(h_out, out, sizeof(T)*N*N*N, hipMemcpyDeviceToHost);
			check_error("Failed to copy device memory to host\n");
			double gups = double(N)*N*N*0.000001 / ftime*ITERATIVE_COUNT;
			const int flop = 25;
			double flops = gups * flop;

			printf("dtype=%s, all_time=%fms, one_itr_time=%fms, N=%d,Iterative=%d, CacheCount=%d, WarpCount=%d,  gups=%.2fGUPS, gflop=%.2fGFLOP/s\n", 
				sizeof(T)==8?"double":"float", ftime, ftime/ ITERATIVE_COUNT, N, ITERATIVE_COUNT, CACHE_DATA_COUNT, WARP_COUNT, gups, flops);
		}
	}

	hipFree(in);
	hipFree(out);
}

template<typename T, int _N> static 
void j3d13pt_gold(const T* l_in, T* l_out, int N) {
	const T(*in)[_N][_N] = (const T(*)[_N][_N])l_in;
	T(*out)[_N][_N] = (T(*)[_N][_N])l_out;

	#pragma omp parallel for
	for (int k = 2; k < N - 2; k++) {
		for (int j = 2; j < N - 2; j++) {
			for (int i = 2; i < N - 2; i++) {
				out[k][j][i] =
					_F3d13PT_022*in[k - 2][j][i] +
					_F3d13PT_122*in[k - 1][j][i] +
					_F3d13PT_222*in[k][j][i] +
					_F3d13PT_322*in[k + 1][j][i] +
					_F3d13PT_422*in[k + 2][j][i] +
					_F3d13PT_212*in[k][j - 1][i] +
					_F3d13PT_202*in[k][j - 2][i] +
					_F3d13PT_232*in[k][j + 1][i] +
					_F3d13PT_242*in[k][j + 2][i] +
					_F3d13PT_220*in[k][j][i - 2] +
					_F3d13PT_221*in[k][j][i - 1] +
					_F3d13PT_223*in[k][j][i + 1] +
					_F3d13PT_224*in[k][j][i + 2];
			}
		}
	}
}

template<typename T, int ITERATIVE_COUNT, int N, int BLOCK_SIZE, int CACHE_DATA_COUNT>
static int _3d13pt(int argc, char** argv) {
	printf("%s:%s:Datatype:%d\n", __FILE__, __FUNCTION__, sizeof(T));
	double error = 0;

	T(*input)[N][N] = (T(*)[N][N]) getRandom3DArray<T>(N, N, N);
	T(*output)[N][N] = (T(*)[N][N]) getZero3DArray<T>(N, N, N);
	T(*output_gold)[N][N] = (T(*)[N][N]) getZero3DArray<T>(N, N, N);

	//const int ITERATIVE_COUNT = 3;
	host_code<T, ITERATIVE_COUNT, BLOCK_SIZE, CACHE_DATA_COUNT>((T*)input, (T*)output, N);

	for (int i = 0; i < ITERATIVE_COUNT; i++) {
		j3d13pt_gold<T, N>((T*)input, (T*)output_gold, N);
		if (i < ITERATIVE_COUNT - 1) {
			memcpy(input, output_gold, sizeof(T)*N*N*N);
		}
	}

	error = checkError3D<T, N>(N, N, (T*)output, (T*)output_gold, _FILTER_SIZE/2*ITERATIVE_COUNT, N - _FILTER_SIZE / 2 * ITERATIVE_COUNT, _FILTER_SIZE / 2 * ITERATIVE_COUNT, N - _FILTER_SIZE / 2 * ITERATIVE_COUNT, _FILTER_SIZE / 2 * ITERATIVE_COUNT, N - _FILTER_SIZE / 2 * ITERATIVE_COUNT);
	printf("N=%d, [Test] RMS Error : %e\n", N, error);

	delete[] input;
	delete[] output;
	delete[] output_gold;

	if (error > TOLERANCE)
		return -1;
	return 0;
}



int stencil_3d13pt(int argc, char** argv) {
	const char* pPrecision = argv[2];
	const bool bDouble = strstr(pPrecision, "double") ? true : false;
#if defined(_DEBUG) || defined(DEBUG)
	const int N = 32;
#else
	const int N = GRID_SIZE;
#endif
	if (IsTeslaV100()) {
		printf("use V100 GPU, ");
		const int B = 512;
		const int C = 12;
		//V100, It 3, 14 24
		//{
		//	const int ITERATIVE_COUNT = 2;
		//	_3d13pt<double, ITERATIVE_COUNT, N, B, C>(argc, argv);
		//	_3d13pt<float, ITERATIVE_COUNT, N, 512, 18>(argc, argv);
		//}
		{
			const int ITERATIVE_COUNT = 2;
			if (bDouble ) _3d13pt<double, ITERATIVE_COUNT, N, 32*16, 17>(argc, argv);
			if (!bDouble) _3d13pt<float, ITERATIVE_COUNT, N, 512, 16>(argc, argv);
			//dtype=double, all_time=5.780832ms, one_itr_time=2.890416ms, N=512,Iterative=2, CacheCount=16, WarpCount=16,  gups=46.44GUPS, gflop=1160.89GFLOP/s
			//dtype=float, all_time=2.700896ms, one_itr_time=1.350448ms, N=512,Iterative=2, CacheCount=16, WarpCount=16,  gups=99.39GUPS, gflop=2484.69GFLOP/s
		}
	} else {
		printf("use P100 and other GPUs, ");
		//For P100 and other GPUs
		const int B = 512;
		const int C = 12;
		//P100  double, It 1, 512, 16
		//P100, float, It 2, 512 20
		//{
		//	const int ITERATIVE_COUNT = 2;
		//	_3d13pt<double, ITERATIVE_COUNT, N, B, C>(argc, argv);
		//	_3d13pt<float, ITERATIVE_COUNT, N, 512, 18>(argc, argv);
		//}
		{
			{
				const int ITERATIVE_COUNT = 1;
				//_3d13pt<double, ITERATIVE_COUNT, N, 512, 16>(argc, argv);
				//_3d13pt<float, ITERATIVE_COUNT, N, 512, 16>(argc, argv);
			}
			{
				const int ITERATIVE_COUNT = 2;
				if (bDouble) _3d13pt<double, ITERATIVE_COUNT, N, 512, 16>(argc, argv);
				if (!bDouble) _3d13pt<float, ITERATIVE_COUNT, N, 512, 20>(argc, argv);
			}
			{
				const int ITERATIVE_COUNT = 2;
				//_3d13pt<double, ITERATIVE_COUNT, N, 512, 16>(argc, argv);
				//_3d13pt<float, ITERATIVE_COUNT, N, 512, 24>(argc, argv);
			}
			//dtype=double, all_time=10.698720ms, one_itr_time=5.349360ms, N=512,Iterative=2, CacheCount=16, WarpCount=16,  gups=25.09GUPS, gflop=627.26GFLOP/s
			//dtype=float, all_time=4.416096ms, one_itr_time=2.208048ms, N=512,Iterative=2, CacheCount=20, WarpCount=16,  gups=60.79GUPS, gflop=1519.64GFLOP/s
			//dtype = double, all_time = 5.399296ms, one_itr_time = 5.399296ms, N = 512, Iterative = 1, CacheCount = 16, WarpCount = 16, gups = 24.86GUPS, gflop = 621.46GFLOP / s
			//dtype=double, all_time=10.387712ms, one_itr_time=5.193856ms, N=512,Iterative=2, CacheCount=17, WarpCount=16,  gups=25.84GUPS, gflop=646.04GFLOP/s
			//dtype=float, all_time=5.131840ms, one_itr_time=2.565920ms, N=512,Iterative=2, CacheCount=18, WarpCount=16,  gups=52.31GUPS, gflop=1307.70GFLOP/s
			//dtype = float, all_time = 5.049408ms, one_itr_time = 2.524704ms, N = 512, Iterative = 2, CacheCount = 16, WarpCount = 16, gups = 53.16GUPS, gflop = 1329.04GFLOP / s

		}
	} 

	return 0;
}
