#include "hip/hip_runtime.h"
#include "../ssai-2dconv/common.h"
#include "../ssai-2dconv/cudaLib.cuh"

#define    _F00         ((T)0.112)   //1
#define    _F01         ((T)0.224)   //2
#define    _F02         ((T)0.336)   //3
#define    _F03         ((T)0.448)   //4
#define    _F04         ((T)0.56)   //5
#define    _F05         ((T)0.672)   //6
#define    _F06         ((T)0.784)   //7
#define    _F07         ((T)0.896)   //8
#define    _F08         ((T)1.008)   //9
#define    _F09         ((T)1.12)   //10
#define    _F010         ((T)1.232)   //11
#define    _F10         ((T)1.344)   //12
#define    _F11         ((T)1.456)   //13
#define    _F12         ((T)1.568)   //14
#define    _F13         ((T)1.68)   //15
#define    _F14         ((T)1.792)   //16
#define    _F15         ((T)1.904)   //17
#define    _F16         ((T)2.016)   //18
#define    _F17         ((T)2.128)   //19
#define    _F18         ((T)2.24)   //20
#define    _F19         ((T)2.352)   //21
#define    _F110         ((T)2.464)   //22
#define    _F20         ((T)2.576)   //23
#define    _F21         ((T)2.688)   //24
#define    _F22         ((T)2.8)   //25
#define    _F23         ((T)2.912)   //26
#define    _F24         ((T)3.024)   //27
#define    _F25         ((T)3.136)   //28
#define    _F26         ((T)3.248)   //29
#define    _F27         ((T)3.36)   //30
#define    _F28         ((T)3.472)   //31
#define    _F29         ((T)3.584)   //32
#define    _F210         ((T)3.696)   //33
#define    _F30         ((T)3.808)   //34
#define    _F31         ((T)3.92)   //35
#define    _F32         ((T)4.032)   //36
#define    _F33         ((T)4.144)   //37
#define    _F34         ((T)4.256)   //38
#define    _F35         ((T)4.368)   //39
#define    _F36         ((T)4.48)   //40
#define    _F37         ((T)4.592)   //41
#define    _F38         ((T)4.704)   //42
#define    _F39         ((T)4.816)   //43
#define    _F310         ((T)4.928)   //44
#define    _F40         ((T)5.04)   //45
#define    _F41         ((T)5.152)   //46
#define    _F42         ((T)5.264)   //47
#define    _F43         ((T)5.376)   //48
#define    _F44         ((T)5.488)   //49
#define    _F45         ((T)5.6)   //50
#define    _F46         ((T)5.712)   //51
#define    _F47         ((T)5.824)   //52
#define    _F48         ((T)5.936)   //53
#define    _F49         ((T)6.048)   //54
#define    _F410         ((T)6.16)   //55
#define    _F50         ((T)6.272)   //56
#define    _F51         ((T)6.384)   //57
#define    _F52         ((T)6.496)   //58
#define    _F53         ((T)6.608)   //59
#define    _F54         ((T)6.72)   //60
#define    _F55         ((T)6.832)   //61
#define    _F56         ((T)6.944)   //62
#define    _F57         ((T)7.056)   //63
#define    _F58         ((T)7.168)   //64
#define    _F59         ((T)7.28)   //65
#define    _F510         ((T)7.392)   //66
#define    _F60         ((T)7.504)   //67
#define    _F61         ((T)7.616)   //68
#define    _F62         ((T)7.728)   //69
#define    _F63         ((T)7.84)   //70
#define    _F64         ((T)7.952)   //71
#define    _F65         ((T)8.064)   //72
#define    _F66         ((T)8.176)   //73
#define    _F67         ((T)8.288)   //74
#define    _F68         ((T)8.4)   //75
#define    _F69         ((T)8.512)   //76
#define    _F610         ((T)8.624)   //77
#define    _F70         ((T)8.736)   //78
#define    _F71         ((T)8.848)   //79
#define    _F72         ((T)8.96)   //80
#define    _F73         ((T)9.072)   //81
#define    _F74         ((T)9.184)   //82
#define    _F75         ((T)9.296)   //83
#define    _F76         ((T)9.408)   //84
#define    _F77         ((T)9.52)   //85
#define    _F78         ((T)9.632)   //86
#define    _F79         ((T)9.744)   //87
#define    _F710         ((T)9.856)   //88
#define    _F80         ((T)9.968)   //89
#define    _F81         ((T)10.08)   //90
#define    _F82         ((T)10.192)   //91
#define    _F83         ((T)10.304)   //92
#define    _F84         ((T)10.416)   //93
#define    _F85         ((T)10.528)   //94
#define    _F86         ((T)10.64)   //95
#define    _F87         ((T)10.752)   //96
#define    _F88         ((T)10.864)   //97
#define    _F89         ((T)10.976)   //98
#define    _F810         ((T)11.088)   //99
#define    _F90         ((T)11.2)   //100
#define    _F91         ((T)11.312)   //101
#define    _F92         ((T)11.424)   //102
#define    _F93         ((T)11.536)   //103
#define    _F94         ((T)11.648)   //104
#define    _F95         ((T)11.76)   //105
#define    _F96         ((T)11.872)   //106
#define    _F97         ((T)11.984)   //107
#define    _F98         ((T)12.096)   //108
#define    _F99         ((T)12.208)   //109
#define    _F910         ((T)12.32)   //110
#define    _F100         ((T)12.432)   //111
#define    _F101         ((T)12.544)   //112
#define    _F102         ((T)12.656)   //113
#define    _F103         ((T)12.768)   //114
#define    _F104         ((T)12.88)   //115
#define    _F105         ((T)12.992)   //116
#define    _F106         ((T)13.104)   //117
#define    _F107         ((T)13.216)   //118
#define    _F108         ((T)13.328)   //119
#define    _F109         ((T)13.44)   //120
#define    _F1010         ((T)13.552)   //121

namespace stencil2d_121pt_v2 {
	static const int WARP_SIZE = 32;
	static const int FILTER_WIDTH = 11;
	static const int FILTER_HEIGHT = 11;

	template<typename T, int BLOCK_SIZE, int PROCESS_DATA_COUNT>
	__global__ void j2d121pt(const T* __restrict__ src, T* dst, int width, int height)
	{
		const int WARP_COUNT = BLOCK_SIZE >> 5;
		const int laneId = threadIdx.x & 31;
		const int warpId = threadIdx.x >> 5;
		const int WARP_PROCESS_DATA_COUNT = WARP_SIZE - FILTER_WIDTH + 1;
		const int BLOCK_PROCESS_DATA_COUNT = WARP_PROCESS_DATA_COUNT*WARP_COUNT;
		const int DATA_CACHE_SIZE = PROCESS_DATA_COUNT + FILTER_HEIGHT - 1;

		T data00, data01, data02, data03, data04, data05, data06, data07, data08, data09, data10, data11, data12, data13, data14, data15, data16, data17;
		//T data[DATA_CACHE_SIZE];

		const int process_count = BLOCK_PROCESS_DATA_COUNT*blockIdx.x + WARP_PROCESS_DATA_COUNT*warpId;
		if (process_count >= width)
			return;
		int tidx = process_count + laneId - FILTER_WIDTH / 2;
		int tidy = PROCESS_DATA_COUNT*blockIdx.y - FILTER_HEIGHT / 2;

		{
			int index = width*tidy + tidx;
			if (tidx < 0)            index -= tidx;
			else if (tidx >= width)  index -= tidx - width + 1;
			if (tidy < 0)            index -= tidy*width;
			else if (tidy >= height) index -= (tidy - height + 1)*width;

			int _tidy = tidy;
			data00 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data01 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data02 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data03 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data04 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data05 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data06 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data07 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data08 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data09 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data10 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data11 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data12 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data13 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data14 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data15 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data16 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
			data17 = src[index]; if (_tidy >= 0 && _tidy < height - 1) { index += width; _tidy++; }
//#pragma unroll
//			for (int s = 0; s < DATA_CACHE_SIZE; s++) {
//				int _tidy = tidy + s;
//				data[s] = src[index];
//				if (_tidy >= 0 && _tidy < height - 1) {
//					//data[s] = src[index];
//					index += width;
//				}
//				//else {
//				//	data[s] = 0;
//				//}
//			}
		}
	
		{
			////unroll 0
			{
				T sum = 0;
				sum += data00 * _F010;
				sum += data01 * _F110;
				sum += data02 * _F210;
				sum += data03 * _F310;
				sum += data04 * _F410;
				sum += data05 * _F510;
				sum += data06 * _F610;
				sum += data07 * _F710;
				sum += data08 * _F810;
				sum += data09 * _F910;
				sum += data10 * _F1010;

				sum = __my_shfl_down(sum, 1);
				sum += data00 * _F09;
				sum += data01 * _F19;
				sum += data02 * _F29;
				sum += data03 * _F39;
				sum += data04 * _F49;
				sum += data05 * _F59;
				sum += data06 * _F69;
				sum += data07 * _F79;
				sum += data08 * _F89;
				sum += data09 * _F99;
				sum += data10 * _F109;

				sum = __my_shfl_down(sum, 1);
				sum += data00 * _F08;
				sum += data01 * _F18;
				sum += data02 * _F28;
				sum += data03 * _F38;
				sum += data04 * _F48;
				sum += data05 * _F58;
				sum += data06 * _F68;
				sum += data07 * _F78;
				sum += data08 * _F88;
				sum += data09 * _F98;
				sum += data10 * _F108;

				sum = __my_shfl_down(sum, 1);
				sum += data00 * _F07;
				sum += data01 * _F17;
				sum += data02 * _F27;
				sum += data03 * _F37;
				sum += data04 * _F47;
				sum += data05 * _F57;
				sum += data06 * _F67;
				sum += data07 * _F77;
				sum += data08 * _F87;
				sum += data09 * _F97;
				sum += data10 * _F107;

				sum = __my_shfl_down(sum, 1);
				sum += data00 * _F06;
				sum += data01 * _F16;
				sum += data02 * _F26;
				sum += data03 * _F36;
				sum += data04 * _F46;
				sum += data05 * _F56;
				sum += data06 * _F66;
				sum += data07 * _F76;
				sum += data08 * _F86;
				sum += data09 * _F96;
				sum += data10 * _F106;

				sum = __my_shfl_down(sum, 1);
				sum += data00 * _F05;
				sum += data01 * _F15;
				sum += data02 * _F25;
				sum += data03 * _F35;
				sum += data04 * _F45;
				sum += data05 * _F55;
				sum += data06 * _F65;
				sum += data07 * _F75;
				sum += data08 * _F85;
				sum += data09 * _F95;
				sum += data10 * _F105;

				sum = __my_shfl_down(sum, 1);
				sum += data00 * _F04;
				sum += data01 * _F14;
				sum += data02 * _F24;
				sum += data03 * _F34;
				sum += data04 * _F44;
				sum += data05 * _F54;
				sum += data06 * _F64;
				sum += data07 * _F74;
				sum += data08 * _F84;
				sum += data09 * _F94;
				sum += data10 * _F104;

				sum = __my_shfl_down(sum, 1);
				sum += data00 * _F03;
				sum += data01 * _F13;
				sum += data02 * _F23;
				sum += data03 * _F33;
				sum += data04 * _F43;
				sum += data05 * _F53;
				sum += data06 * _F63;
				sum += data07 * _F73;
				sum += data08 * _F83;
				sum += data09 * _F93;
				sum += data10 * _F103;

				sum = __my_shfl_down(sum, 1);
				sum += data00 * _F02;
				sum += data01 * _F12;
				sum += data02 * _F22;
				sum += data03 * _F32;
				sum += data04 * _F42;
				sum += data05 * _F52;
				sum += data06 * _F62;
				sum += data07 * _F72;
				sum += data08 * _F82;
				sum += data09 * _F92;
				sum += data10 * _F102;

				sum = __my_shfl_down(sum, 1);
				sum += data00 * _F01;
				sum += data01 * _F11;
				sum += data02 * _F21;
				sum += data03 * _F31;
				sum += data04 * _F41;
				sum += data05 * _F51;
				sum += data06 * _F61;
				sum += data07 * _F71;
				sum += data08 * _F81;
				sum += data09 * _F91;
				sum += data10 * _F101;

				sum = __my_shfl_down(sum, 1);
				sum += data00 * _F00;
				sum += data01 * _F10;
				sum += data02 * _F20;
				sum += data03 * _F30;
				sum += data04 * _F40;
				sum += data05 * _F50;
				sum += data06 * _F60;
				sum += data07 * _F70;
				sum += data08 * _F80;
				sum += data09 * _F90;
				sum += data10 * _F100;

				data00 = sum;

			}
			////unroll 1
			{
				T sum = 0;
				sum += data01 * _F010;
				sum += data02 * _F110;
				sum += data03 * _F210;
				sum += data04 * _F310;
				sum += data05 * _F410;
				sum += data06 * _F510;
				sum += data07 * _F610;
				sum += data08 * _F710;
				sum += data09 * _F810;
				sum += data10 * _F910;
				sum += data11 * _F1010;

				sum = __my_shfl_down(sum, 1);
				sum += data01 * _F09;
				sum += data02 * _F19;
				sum += data03 * _F29;
				sum += data04 * _F39;
				sum += data05 * _F49;
				sum += data06 * _F59;
				sum += data07 * _F69;
				sum += data08 * _F79;
				sum += data09 * _F89;
				sum += data10 * _F99;
				sum += data11 * _F109;

				sum = __my_shfl_down(sum, 1);
				sum += data01 * _F08;
				sum += data02 * _F18;
				sum += data03 * _F28;
				sum += data04 * _F38;
				sum += data05 * _F48;
				sum += data06 * _F58;
				sum += data07 * _F68;
				sum += data08 * _F78;
				sum += data09 * _F88;
				sum += data10 * _F98;
				sum += data11 * _F108;

				sum = __my_shfl_down(sum, 1);
				sum += data01 * _F07;
				sum += data02 * _F17;
				sum += data03 * _F27;
				sum += data04 * _F37;
				sum += data05 * _F47;
				sum += data06 * _F57;
				sum += data07 * _F67;
				sum += data08 * _F77;
				sum += data09 * _F87;
				sum += data10 * _F97;
				sum += data11 * _F107;

				sum = __my_shfl_down(sum, 1);
				sum += data01 * _F06;
				sum += data02 * _F16;
				sum += data03 * _F26;
				sum += data04 * _F36;
				sum += data05 * _F46;
				sum += data06 * _F56;
				sum += data07 * _F66;
				sum += data08 * _F76;
				sum += data09 * _F86;
				sum += data10 * _F96;
				sum += data11 * _F106;

				sum = __my_shfl_down(sum, 1);
				sum += data01 * _F05;
				sum += data02 * _F15;
				sum += data03 * _F25;
				sum += data04 * _F35;
				sum += data05 * _F45;
				sum += data06 * _F55;
				sum += data07 * _F65;
				sum += data08 * _F75;
				sum += data09 * _F85;
				sum += data10 * _F95;
				sum += data11 * _F105;

				sum = __my_shfl_down(sum, 1);
				sum += data01 * _F04;
				sum += data02 * _F14;
				sum += data03 * _F24;
				sum += data04 * _F34;
				sum += data05 * _F44;
				sum += data06 * _F54;
				sum += data07 * _F64;
				sum += data08 * _F74;
				sum += data09 * _F84;
				sum += data10 * _F94;
				sum += data11 * _F104;

				sum = __my_shfl_down(sum, 1);
				sum += data01 * _F03;
				sum += data02 * _F13;
				sum += data03 * _F23;
				sum += data04 * _F33;
				sum += data05 * _F43;
				sum += data06 * _F53;
				sum += data07 * _F63;
				sum += data08 * _F73;
				sum += data09 * _F83;
				sum += data10 * _F93;
				sum += data11 * _F103;

				sum = __my_shfl_down(sum, 1);
				sum += data01 * _F02;
				sum += data02 * _F12;
				sum += data03 * _F22;
				sum += data04 * _F32;
				sum += data05 * _F42;
				sum += data06 * _F52;
				sum += data07 * _F62;
				sum += data08 * _F72;
				sum += data09 * _F82;
				sum += data10 * _F92;
				sum += data11 * _F102;

				sum = __my_shfl_down(sum, 1);
				sum += data01 * _F01;
				sum += data02 * _F11;
				sum += data03 * _F21;
				sum += data04 * _F31;
				sum += data05 * _F41;
				sum += data06 * _F51;
				sum += data07 * _F61;
				sum += data08 * _F71;
				sum += data09 * _F81;
				sum += data10 * _F91;
				sum += data11 * _F101;

				sum = __my_shfl_down(sum, 1);
				sum += data01 * _F00;
				sum += data02 * _F10;
				sum += data03 * _F20;
				sum += data04 * _F30;
				sum += data05 * _F40;
				sum += data06 * _F50;
				sum += data07 * _F60;
				sum += data08 * _F70;
				sum += data09 * _F80;
				sum += data10 * _F90;
				sum += data11 * _F100;

				data01 = sum;

			}
			////unroll 2
			{
				T sum = 0;
				sum += data02 * _F010;
				sum += data03 * _F110;
				sum += data04 * _F210;
				sum += data05 * _F310;
				sum += data06 * _F410;
				sum += data07 * _F510;
				sum += data08 * _F610;
				sum += data09 * _F710;
				sum += data10 * _F810;
				sum += data11 * _F910;
				sum += data12 * _F1010;

				sum = __my_shfl_down(sum, 1);
				sum += data02 * _F09;
				sum += data03 * _F19;
				sum += data04 * _F29;
				sum += data05 * _F39;
				sum += data06 * _F49;
				sum += data07 * _F59;
				sum += data08 * _F69;
				sum += data09 * _F79;
				sum += data10 * _F89;
				sum += data11 * _F99;
				sum += data12 * _F109;

				sum = __my_shfl_down(sum, 1);
				sum += data02 * _F08;
				sum += data03 * _F18;
				sum += data04 * _F28;
				sum += data05 * _F38;
				sum += data06 * _F48;
				sum += data07 * _F58;
				sum += data08 * _F68;
				sum += data09 * _F78;
				sum += data10 * _F88;
				sum += data11 * _F98;
				sum += data12 * _F108;

				sum = __my_shfl_down(sum, 1);
				sum += data02 * _F07;
				sum += data03 * _F17;
				sum += data04 * _F27;
				sum += data05 * _F37;
				sum += data06 * _F47;
				sum += data07 * _F57;
				sum += data08 * _F67;
				sum += data09 * _F77;
				sum += data10 * _F87;
				sum += data11 * _F97;
				sum += data12 * _F107;

				sum = __my_shfl_down(sum, 1);
				sum += data02 * _F06;
				sum += data03 * _F16;
				sum += data04 * _F26;
				sum += data05 * _F36;
				sum += data06 * _F46;
				sum += data07 * _F56;
				sum += data08 * _F66;
				sum += data09 * _F76;
				sum += data10 * _F86;
				sum += data11 * _F96;
				sum += data12 * _F106;

				sum = __my_shfl_down(sum, 1);
				sum += data02 * _F05;
				sum += data03 * _F15;
				sum += data04 * _F25;
				sum += data05 * _F35;
				sum += data06 * _F45;
				sum += data07 * _F55;
				sum += data08 * _F65;
				sum += data09 * _F75;
				sum += data10 * _F85;
				sum += data11 * _F95;
				sum += data12 * _F105;

				sum = __my_shfl_down(sum, 1);
				sum += data02 * _F04;
				sum += data03 * _F14;
				sum += data04 * _F24;
				sum += data05 * _F34;
				sum += data06 * _F44;
				sum += data07 * _F54;
				sum += data08 * _F64;
				sum += data09 * _F74;
				sum += data10 * _F84;
				sum += data11 * _F94;
				sum += data12 * _F104;

				sum = __my_shfl_down(sum, 1);
				sum += data02 * _F03;
				sum += data03 * _F13;
				sum += data04 * _F23;
				sum += data05 * _F33;
				sum += data06 * _F43;
				sum += data07 * _F53;
				sum += data08 * _F63;
				sum += data09 * _F73;
				sum += data10 * _F83;
				sum += data11 * _F93;
				sum += data12 * _F103;

				sum = __my_shfl_down(sum, 1);
				sum += data02 * _F02;
				sum += data03 * _F12;
				sum += data04 * _F22;
				sum += data05 * _F32;
				sum += data06 * _F42;
				sum += data07 * _F52;
				sum += data08 * _F62;
				sum += data09 * _F72;
				sum += data10 * _F82;
				sum += data11 * _F92;
				sum += data12 * _F102;

				sum = __my_shfl_down(sum, 1);
				sum += data02 * _F01;
				sum += data03 * _F11;
				sum += data04 * _F21;
				sum += data05 * _F31;
				sum += data06 * _F41;
				sum += data07 * _F51;
				sum += data08 * _F61;
				sum += data09 * _F71;
				sum += data10 * _F81;
				sum += data11 * _F91;
				sum += data12 * _F101;

				sum = __my_shfl_down(sum, 1);
				sum += data02 * _F00;
				sum += data03 * _F10;
				sum += data04 * _F20;
				sum += data05 * _F30;
				sum += data06 * _F40;
				sum += data07 * _F50;
				sum += data08 * _F60;
				sum += data09 * _F70;
				sum += data10 * _F80;
				sum += data11 * _F90;
				sum += data12 * _F100;

				data02 = sum;

			}
			////unroll 3
			{
				T sum = 0;
				sum += data03 * _F010;
				sum += data04 * _F110;
				sum += data05 * _F210;
				sum += data06 * _F310;
				sum += data07 * _F410;
				sum += data08 * _F510;
				sum += data09 * _F610;
				sum += data10 * _F710;
				sum += data11 * _F810;
				sum += data12 * _F910;
				sum += data13 * _F1010;

				sum = __my_shfl_down(sum, 1);
				sum += data03 * _F09;
				sum += data04 * _F19;
				sum += data05 * _F29;
				sum += data06 * _F39;
				sum += data07 * _F49;
				sum += data08 * _F59;
				sum += data09 * _F69;
				sum += data10 * _F79;
				sum += data11 * _F89;
				sum += data12 * _F99;
				sum += data13 * _F109;

				sum = __my_shfl_down(sum, 1);
				sum += data03 * _F08;
				sum += data04 * _F18;
				sum += data05 * _F28;
				sum += data06 * _F38;
				sum += data07 * _F48;
				sum += data08 * _F58;
				sum += data09 * _F68;
				sum += data10 * _F78;
				sum += data11 * _F88;
				sum += data12 * _F98;
				sum += data13 * _F108;

				sum = __my_shfl_down(sum, 1);
				sum += data03 * _F07;
				sum += data04 * _F17;
				sum += data05 * _F27;
				sum += data06 * _F37;
				sum += data07 * _F47;
				sum += data08 * _F57;
				sum += data09 * _F67;
				sum += data10 * _F77;
				sum += data11 * _F87;
				sum += data12 * _F97;
				sum += data13 * _F107;

				sum = __my_shfl_down(sum, 1);
				sum += data03 * _F06;
				sum += data04 * _F16;
				sum += data05 * _F26;
				sum += data06 * _F36;
				sum += data07 * _F46;
				sum += data08 * _F56;
				sum += data09 * _F66;
				sum += data10 * _F76;
				sum += data11 * _F86;
				sum += data12 * _F96;
				sum += data13 * _F106;

				sum = __my_shfl_down(sum, 1);
				sum += data03 * _F05;
				sum += data04 * _F15;
				sum += data05 * _F25;
				sum += data06 * _F35;
				sum += data07 * _F45;
				sum += data08 * _F55;
				sum += data09 * _F65;
				sum += data10 * _F75;
				sum += data11 * _F85;
				sum += data12 * _F95;
				sum += data13 * _F105;

				sum = __my_shfl_down(sum, 1);
				sum += data03 * _F04;
				sum += data04 * _F14;
				sum += data05 * _F24;
				sum += data06 * _F34;
				sum += data07 * _F44;
				sum += data08 * _F54;
				sum += data09 * _F64;
				sum += data10 * _F74;
				sum += data11 * _F84;
				sum += data12 * _F94;
				sum += data13 * _F104;

				sum = __my_shfl_down(sum, 1);
				sum += data03 * _F03;
				sum += data04 * _F13;
				sum += data05 * _F23;
				sum += data06 * _F33;
				sum += data07 * _F43;
				sum += data08 * _F53;
				sum += data09 * _F63;
				sum += data10 * _F73;
				sum += data11 * _F83;
				sum += data12 * _F93;
				sum += data13 * _F103;

				sum = __my_shfl_down(sum, 1);
				sum += data03 * _F02;
				sum += data04 * _F12;
				sum += data05 * _F22;
				sum += data06 * _F32;
				sum += data07 * _F42;
				sum += data08 * _F52;
				sum += data09 * _F62;
				sum += data10 * _F72;
				sum += data11 * _F82;
				sum += data12 * _F92;
				sum += data13 * _F102;

				sum = __my_shfl_down(sum, 1);
				sum += data03 * _F01;
				sum += data04 * _F11;
				sum += data05 * _F21;
				sum += data06 * _F31;
				sum += data07 * _F41;
				sum += data08 * _F51;
				sum += data09 * _F61;
				sum += data10 * _F71;
				sum += data11 * _F81;
				sum += data12 * _F91;
				sum += data13 * _F101;

				sum = __my_shfl_down(sum, 1);
				sum += data03 * _F00;
				sum += data04 * _F10;
				sum += data05 * _F20;
				sum += data06 * _F30;
				sum += data07 * _F40;
				sum += data08 * _F50;
				sum += data09 * _F60;
				sum += data10 * _F70;
				sum += data11 * _F80;
				sum += data12 * _F90;
				sum += data13 * _F100;

				data03 = sum;

			}
			////unroll 4
			{
				T sum = 0;
				sum += data04 * _F010;
				sum += data05 * _F110;
				sum += data06 * _F210;
				sum += data07 * _F310;
				sum += data08 * _F410;
				sum += data09 * _F510;
				sum += data10 * _F610;
				sum += data11 * _F710;
				sum += data12 * _F810;
				sum += data13 * _F910;
				sum += data14 * _F1010;

				sum = __my_shfl_down(sum, 1);
				sum += data04 * _F09;
				sum += data05 * _F19;
				sum += data06 * _F29;
				sum += data07 * _F39;
				sum += data08 * _F49;
				sum += data09 * _F59;
				sum += data10 * _F69;
				sum += data11 * _F79;
				sum += data12 * _F89;
				sum += data13 * _F99;
				sum += data14 * _F109;

				sum = __my_shfl_down(sum, 1);
				sum += data04 * _F08;
				sum += data05 * _F18;
				sum += data06 * _F28;
				sum += data07 * _F38;
				sum += data08 * _F48;
				sum += data09 * _F58;
				sum += data10 * _F68;
				sum += data11 * _F78;
				sum += data12 * _F88;
				sum += data13 * _F98;
				sum += data14 * _F108;

				sum = __my_shfl_down(sum, 1);
				sum += data04 * _F07;
				sum += data05 * _F17;
				sum += data06 * _F27;
				sum += data07 * _F37;
				sum += data08 * _F47;
				sum += data09 * _F57;
				sum += data10 * _F67;
				sum += data11 * _F77;
				sum += data12 * _F87;
				sum += data13 * _F97;
				sum += data14 * _F107;

				sum = __my_shfl_down(sum, 1);
				sum += data04 * _F06;
				sum += data05 * _F16;
				sum += data06 * _F26;
				sum += data07 * _F36;
				sum += data08 * _F46;
				sum += data09 * _F56;
				sum += data10 * _F66;
				sum += data11 * _F76;
				sum += data12 * _F86;
				sum += data13 * _F96;
				sum += data14 * _F106;

				sum = __my_shfl_down(sum, 1);
				sum += data04 * _F05;
				sum += data05 * _F15;
				sum += data06 * _F25;
				sum += data07 * _F35;
				sum += data08 * _F45;
				sum += data09 * _F55;
				sum += data10 * _F65;
				sum += data11 * _F75;
				sum += data12 * _F85;
				sum += data13 * _F95;
				sum += data14 * _F105;

				sum = __my_shfl_down(sum, 1);
				sum += data04 * _F04;
				sum += data05 * _F14;
				sum += data06 * _F24;
				sum += data07 * _F34;
				sum += data08 * _F44;
				sum += data09 * _F54;
				sum += data10 * _F64;
				sum += data11 * _F74;
				sum += data12 * _F84;
				sum += data13 * _F94;
				sum += data14 * _F104;

				sum = __my_shfl_down(sum, 1);
				sum += data04 * _F03;
				sum += data05 * _F13;
				sum += data06 * _F23;
				sum += data07 * _F33;
				sum += data08 * _F43;
				sum += data09 * _F53;
				sum += data10 * _F63;
				sum += data11 * _F73;
				sum += data12 * _F83;
				sum += data13 * _F93;
				sum += data14 * _F103;

				sum = __my_shfl_down(sum, 1);
				sum += data04 * _F02;
				sum += data05 * _F12;
				sum += data06 * _F22;
				sum += data07 * _F32;
				sum += data08 * _F42;
				sum += data09 * _F52;
				sum += data10 * _F62;
				sum += data11 * _F72;
				sum += data12 * _F82;
				sum += data13 * _F92;
				sum += data14 * _F102;

				sum = __my_shfl_down(sum, 1);
				sum += data04 * _F01;
				sum += data05 * _F11;
				sum += data06 * _F21;
				sum += data07 * _F31;
				sum += data08 * _F41;
				sum += data09 * _F51;
				sum += data10 * _F61;
				sum += data11 * _F71;
				sum += data12 * _F81;
				sum += data13 * _F91;
				sum += data14 * _F101;

				sum = __my_shfl_down(sum, 1);
				sum += data04 * _F00;
				sum += data05 * _F10;
				sum += data06 * _F20;
				sum += data07 * _F30;
				sum += data08 * _F40;
				sum += data09 * _F50;
				sum += data10 * _F60;
				sum += data11 * _F70;
				sum += data12 * _F80;
				sum += data13 * _F90;
				sum += data14 * _F100;

				data04 = sum;

			}
			////unroll 5
			{
				T sum = 0;
				sum += data05 * _F010;
				sum += data06 * _F110;
				sum += data07 * _F210;
				sum += data08 * _F310;
				sum += data09 * _F410;
				sum += data10 * _F510;
				sum += data11 * _F610;
				sum += data12 * _F710;
				sum += data13 * _F810;
				sum += data14 * _F910;
				sum += data15 * _F1010;

				sum = __my_shfl_down(sum, 1);
				sum += data05 * _F09;
				sum += data06 * _F19;
				sum += data07 * _F29;
				sum += data08 * _F39;
				sum += data09 * _F49;
				sum += data10 * _F59;
				sum += data11 * _F69;
				sum += data12 * _F79;
				sum += data13 * _F89;
				sum += data14 * _F99;
				sum += data15 * _F109;

				sum = __my_shfl_down(sum, 1);
				sum += data05 * _F08;
				sum += data06 * _F18;
				sum += data07 * _F28;
				sum += data08 * _F38;
				sum += data09 * _F48;
				sum += data10 * _F58;
				sum += data11 * _F68;
				sum += data12 * _F78;
				sum += data13 * _F88;
				sum += data14 * _F98;
				sum += data15 * _F108;

				sum = __my_shfl_down(sum, 1);
				sum += data05 * _F07;
				sum += data06 * _F17;
				sum += data07 * _F27;
				sum += data08 * _F37;
				sum += data09 * _F47;
				sum += data10 * _F57;
				sum += data11 * _F67;
				sum += data12 * _F77;
				sum += data13 * _F87;
				sum += data14 * _F97;
				sum += data15 * _F107;

				sum = __my_shfl_down(sum, 1);
				sum += data05 * _F06;
				sum += data06 * _F16;
				sum += data07 * _F26;
				sum += data08 * _F36;
				sum += data09 * _F46;
				sum += data10 * _F56;
				sum += data11 * _F66;
				sum += data12 * _F76;
				sum += data13 * _F86;
				sum += data14 * _F96;
				sum += data15 * _F106;

				sum = __my_shfl_down(sum, 1);
				sum += data05 * _F05;
				sum += data06 * _F15;
				sum += data07 * _F25;
				sum += data08 * _F35;
				sum += data09 * _F45;
				sum += data10 * _F55;
				sum += data11 * _F65;
				sum += data12 * _F75;
				sum += data13 * _F85;
				sum += data14 * _F95;
				sum += data15 * _F105;

				sum = __my_shfl_down(sum, 1);
				sum += data05 * _F04;
				sum += data06 * _F14;
				sum += data07 * _F24;
				sum += data08 * _F34;
				sum += data09 * _F44;
				sum += data10 * _F54;
				sum += data11 * _F64;
				sum += data12 * _F74;
				sum += data13 * _F84;
				sum += data14 * _F94;
				sum += data15 * _F104;

				sum = __my_shfl_down(sum, 1);
				sum += data05 * _F03;
				sum += data06 * _F13;
				sum += data07 * _F23;
				sum += data08 * _F33;
				sum += data09 * _F43;
				sum += data10 * _F53;
				sum += data11 * _F63;
				sum += data12 * _F73;
				sum += data13 * _F83;
				sum += data14 * _F93;
				sum += data15 * _F103;

				sum = __my_shfl_down(sum, 1);
				sum += data05 * _F02;
				sum += data06 * _F12;
				sum += data07 * _F22;
				sum += data08 * _F32;
				sum += data09 * _F42;
				sum += data10 * _F52;
				sum += data11 * _F62;
				sum += data12 * _F72;
				sum += data13 * _F82;
				sum += data14 * _F92;
				sum += data15 * _F102;

				sum = __my_shfl_down(sum, 1);
				sum += data05 * _F01;
				sum += data06 * _F11;
				sum += data07 * _F21;
				sum += data08 * _F31;
				sum += data09 * _F41;
				sum += data10 * _F51;
				sum += data11 * _F61;
				sum += data12 * _F71;
				sum += data13 * _F81;
				sum += data14 * _F91;
				sum += data15 * _F101;

				sum = __my_shfl_down(sum, 1);
				sum += data05 * _F00;
				sum += data06 * _F10;
				sum += data07 * _F20;
				sum += data08 * _F30;
				sum += data09 * _F40;
				sum += data10 * _F50;
				sum += data11 * _F60;
				sum += data12 * _F70;
				sum += data13 * _F80;
				sum += data14 * _F90;
				sum += data15 * _F100;

				data05 = sum;

			}
			////unroll 6
			{
				T sum = 0;
				sum += data06 * _F010;
				sum += data07 * _F110;
				sum += data08 * _F210;
				sum += data09 * _F310;
				sum += data10 * _F410;
				sum += data11 * _F510;
				sum += data12 * _F610;
				sum += data13 * _F710;
				sum += data14 * _F810;
				sum += data15 * _F910;
				sum += data16 * _F1010;

				sum = __my_shfl_down(sum, 1);
				sum += data06 * _F09;
				sum += data07 * _F19;
				sum += data08 * _F29;
				sum += data09 * _F39;
				sum += data10 * _F49;
				sum += data11 * _F59;
				sum += data12 * _F69;
				sum += data13 * _F79;
				sum += data14 * _F89;
				sum += data15 * _F99;
				sum += data16 * _F109;

				sum = __my_shfl_down(sum, 1);
				sum += data06 * _F08;
				sum += data07 * _F18;
				sum += data08 * _F28;
				sum += data09 * _F38;
				sum += data10 * _F48;
				sum += data11 * _F58;
				sum += data12 * _F68;
				sum += data13 * _F78;
				sum += data14 * _F88;
				sum += data15 * _F98;
				sum += data16 * _F108;

				sum = __my_shfl_down(sum, 1);
				sum += data06 * _F07;
				sum += data07 * _F17;
				sum += data08 * _F27;
				sum += data09 * _F37;
				sum += data10 * _F47;
				sum += data11 * _F57;
				sum += data12 * _F67;
				sum += data13 * _F77;
				sum += data14 * _F87;
				sum += data15 * _F97;
				sum += data16 * _F107;

				sum = __my_shfl_down(sum, 1);
				sum += data06 * _F06;
				sum += data07 * _F16;
				sum += data08 * _F26;
				sum += data09 * _F36;
				sum += data10 * _F46;
				sum += data11 * _F56;
				sum += data12 * _F66;
				sum += data13 * _F76;
				sum += data14 * _F86;
				sum += data15 * _F96;
				sum += data16 * _F106;

				sum = __my_shfl_down(sum, 1);
				sum += data06 * _F05;
				sum += data07 * _F15;
				sum += data08 * _F25;
				sum += data09 * _F35;
				sum += data10 * _F45;
				sum += data11 * _F55;
				sum += data12 * _F65;
				sum += data13 * _F75;
				sum += data14 * _F85;
				sum += data15 * _F95;
				sum += data16 * _F105;

				sum = __my_shfl_down(sum, 1);
				sum += data06 * _F04;
				sum += data07 * _F14;
				sum += data08 * _F24;
				sum += data09 * _F34;
				sum += data10 * _F44;
				sum += data11 * _F54;
				sum += data12 * _F64;
				sum += data13 * _F74;
				sum += data14 * _F84;
				sum += data15 * _F94;
				sum += data16 * _F104;

				sum = __my_shfl_down(sum, 1);
				sum += data06 * _F03;
				sum += data07 * _F13;
				sum += data08 * _F23;
				sum += data09 * _F33;
				sum += data10 * _F43;
				sum += data11 * _F53;
				sum += data12 * _F63;
				sum += data13 * _F73;
				sum += data14 * _F83;
				sum += data15 * _F93;
				sum += data16 * _F103;

				sum = __my_shfl_down(sum, 1);
				sum += data06 * _F02;
				sum += data07 * _F12;
				sum += data08 * _F22;
				sum += data09 * _F32;
				sum += data10 * _F42;
				sum += data11 * _F52;
				sum += data12 * _F62;
				sum += data13 * _F72;
				sum += data14 * _F82;
				sum += data15 * _F92;
				sum += data16 * _F102;

				sum = __my_shfl_down(sum, 1);
				sum += data06 * _F01;
				sum += data07 * _F11;
				sum += data08 * _F21;
				sum += data09 * _F31;
				sum += data10 * _F41;
				sum += data11 * _F51;
				sum += data12 * _F61;
				sum += data13 * _F71;
				sum += data14 * _F81;
				sum += data15 * _F91;
				sum += data16 * _F101;

				sum = __my_shfl_down(sum, 1);
				sum += data06 * _F00;
				sum += data07 * _F10;
				sum += data08 * _F20;
				sum += data09 * _F30;
				sum += data10 * _F40;
				sum += data11 * _F50;
				sum += data12 * _F60;
				sum += data13 * _F70;
				sum += data14 * _F80;
				sum += data15 * _F90;
				sum += data16 * _F100;

				data06 = sum;

			}
			////unroll 7
			{
				T sum = 0;
				sum += data07 * _F010;
				sum += data08 * _F110;
				sum += data09 * _F210;
				sum += data10 * _F310;
				sum += data11 * _F410;
				sum += data12 * _F510;
				sum += data13 * _F610;
				sum += data14 * _F710;
				sum += data15 * _F810;
				sum += data16 * _F910;
				sum += data17 * _F1010;

				sum = __my_shfl_down(sum, 1);
				sum += data07 * _F09;
				sum += data08 * _F19;
				sum += data09 * _F29;
				sum += data10 * _F39;
				sum += data11 * _F49;
				sum += data12 * _F59;
				sum += data13 * _F69;
				sum += data14 * _F79;
				sum += data15 * _F89;
				sum += data16 * _F99;
				sum += data17 * _F109;

				sum = __my_shfl_down(sum, 1);
				sum += data07 * _F08;
				sum += data08 * _F18;
				sum += data09 * _F28;
				sum += data10 * _F38;
				sum += data11 * _F48;
				sum += data12 * _F58;
				sum += data13 * _F68;
				sum += data14 * _F78;
				sum += data15 * _F88;
				sum += data16 * _F98;
				sum += data17 * _F108;

				sum = __my_shfl_down(sum, 1);
				sum += data07 * _F07;
				sum += data08 * _F17;
				sum += data09 * _F27;
				sum += data10 * _F37;
				sum += data11 * _F47;
				sum += data12 * _F57;
				sum += data13 * _F67;
				sum += data14 * _F77;
				sum += data15 * _F87;
				sum += data16 * _F97;
				sum += data17 * _F107;

				sum = __my_shfl_down(sum, 1);
				sum += data07 * _F06;
				sum += data08 * _F16;
				sum += data09 * _F26;
				sum += data10 * _F36;
				sum += data11 * _F46;
				sum += data12 * _F56;
				sum += data13 * _F66;
				sum += data14 * _F76;
				sum += data15 * _F86;
				sum += data16 * _F96;
				sum += data17 * _F106;

				sum = __my_shfl_down(sum, 1);
				sum += data07 * _F05;
				sum += data08 * _F15;
				sum += data09 * _F25;
				sum += data10 * _F35;
				sum += data11 * _F45;
				sum += data12 * _F55;
				sum += data13 * _F65;
				sum += data14 * _F75;
				sum += data15 * _F85;
				sum += data16 * _F95;
				sum += data17 * _F105;

				sum = __my_shfl_down(sum, 1);
				sum += data07 * _F04;
				sum += data08 * _F14;
				sum += data09 * _F24;
				sum += data10 * _F34;
				sum += data11 * _F44;
				sum += data12 * _F54;
				sum += data13 * _F64;
				sum += data14 * _F74;
				sum += data15 * _F84;
				sum += data16 * _F94;
				sum += data17 * _F104;

				sum = __my_shfl_down(sum, 1);
				sum += data07 * _F03;
				sum += data08 * _F13;
				sum += data09 * _F23;
				sum += data10 * _F33;
				sum += data11 * _F43;
				sum += data12 * _F53;
				sum += data13 * _F63;
				sum += data14 * _F73;
				sum += data15 * _F83;
				sum += data16 * _F93;
				sum += data17 * _F103;

				sum = __my_shfl_down(sum, 1);
				sum += data07 * _F02;
				sum += data08 * _F12;
				sum += data09 * _F22;
				sum += data10 * _F32;
				sum += data11 * _F42;
				sum += data12 * _F52;
				sum += data13 * _F62;
				sum += data14 * _F72;
				sum += data15 * _F82;
				sum += data16 * _F92;
				sum += data17 * _F102;

				sum = __my_shfl_down(sum, 1);
				sum += data07 * _F01;
				sum += data08 * _F11;
				sum += data09 * _F21;
				sum += data10 * _F31;
				sum += data11 * _F41;
				sum += data12 * _F51;
				sum += data13 * _F61;
				sum += data14 * _F71;
				sum += data15 * _F81;
				sum += data16 * _F91;
				sum += data17 * _F101;

				sum = __my_shfl_down(sum, 1);
				sum += data07 * _F00;
				sum += data08 * _F10;
				sum += data09 * _F20;
				sum += data10 * _F30;
				sum += data11 * _F40;
				sum += data12 * _F50;
				sum += data13 * _F60;
				sum += data14 * _F70;
				sum += data15 * _F80;
				sum += data16 * _F90;
				sum += data17 * _F100;

				data07 = sum;
			}
		}

		if (laneId >= WARP_SIZE - FILTER_WIDTH + 1)
			return;

		int _x = tidx + FILTER_WIDTH / 2;
		int _y = tidy + FILTER_HEIGHT / 2;
		int index = width*_y + _x;
		if (_x >= width)
			return;

		if (_y < height) { dst[index] = data00; index += width; _y++; }
		if (_y < height) { dst[index] = data01; index += width; _y++; }
		if (_y < height) { dst[index] = data02; index += width; _y++; }
		if (_y < height) { dst[index] = data03; index += width; _y++; }
		if (_y < height) { dst[index] = data04; index += width; _y++; }
		if (_y < height) { dst[index] = data05; index += width; _y++; }
		if (_y < height) { dst[index] = data06; index += width; _y++; }
		if (_y < height) { dst[index] = data07; index += width; _y++; }
//#pragma unroll
//		for (int i = 0; i < PROCESS_DATA_COUNT; i++) {
//			if (_y + i < height) {
//				dst[index] = data[i];
//				index += width;
//			}
//		}
	}

	template<class DataType, int PROCESS_DATA_COUNT, int BLOCK_SIZE>
	static float Test(int width, int height) {
		const int WARP_COUNT = BLOCK_SIZE >> 5;
		const int WARP_PROCESS_DATA_COUNT = WARP_SIZE - FILTER_WIDTH + 1;
		const int BLOCK_PROCESS_DATA_COUNT = WARP_PROCESS_DATA_COUNT*WARP_COUNT;

		const int nRepeatCount = 1;
		float inc = 0;
		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		//StopWatchWin watch;
		DataT<DataType> img;
		char szPath[1024] = "";
		sprintf(szPath, "../data/Lena%dx%d.raw", width, height);
		bool bRtn = img.Load_uchar(szPath, width, height);
		//sprintf(szPath, "../data/Lena%dx%d.txt", width, height);
		//img.SaveText(szPath);
		if (!bRtn) {
			printf("Load failed : %s, generate random data\n", szPath);
			img.MallocBuffer(width, height);
			for (int i = 0; i < img.width*img.height; i++) {
				img.data[i] = std::rand() % 256;
				//img.data[i] = i/img.width;
			}
		}
		else {
			printf("Load success : %s\n", szPath);
		}

		DevBuffer<DataType> devSrc(width, height), devDst(width, height);
		devSrc.CopyFromHost(img.data, img.width, img.width, img.height);
		DataT<DataType> imgDst;
		imgDst.MallocBuffer(width, height);

		dim3 block_size(BLOCK_SIZE, 1);
		dim3 grid_size(UpDivide(width, BLOCK_PROCESS_DATA_COUNT), UpDivide(height, PROCESS_DATA_COUNT));

		DataType filter[FILTER_HEIGHT][FILTER_WIDTH] =
		{ 
			{ 0.112, 0.224, 0.336, 0.448, 0.56, 0.672, 0.784, 0.896, 1.008, 1.12, 1.232, },
			{ 1.344, 1.456, 1.568, 1.68, 1.792, 1.904, 2.016, 2.128, 2.24, 2.352, 2.464, },
			{ 2.576, 2.688, 2.8, 2.912, 3.024, 3.136, 3.248, 3.36, 3.472, 3.584, 3.696, },
			{ 3.808, 3.92, 4.032, 4.144, 4.256, 4.368, 4.48, 4.592, 4.704, 4.816, 4.928, },
			{ 5.04, 5.152, 5.264, 5.376, 5.488, 5.6, 5.712, 5.824, 5.936, 6.048, 6.16, },
			{ 6.272, 6.384, 6.496, 6.608, 6.72, 6.832, 6.944, 7.056, 7.168, 7.28, 7.392, },
			{ 7.504, 7.616, 7.728, 7.84, 7.952, 8.064, 8.176, 8.288, 8.4, 8.512, 8.624, },
			{ 8.736, 8.848, 8.96, 9.072, 9.184, 9.296, 9.408, 9.52, 9.632, 9.744, 9.856, },
			{ 9.968, 10.08, 10.192, 10.304, 10.416, 10.528, 10.64, 10.752, 10.864, 10.976, 11.088, },
			{ 11.2, 11.312, 11.424, 11.536, 11.648, 11.76, 11.872, 11.984, 12.096, 12.208, 12.32, },
			{ 12.432, 12.544, 12.656, 12.768, 12.88, 12.992, 13.104, 13.216, 13.328, 13.44, 13.552, },
		};

		hipEventRecord(start, 0);
		for (int s = 0; s < nRepeatCount; s++) {
			j2d121pt<DataType, BLOCK_SIZE, PROCESS_DATA_COUNT> <<<grid_size, block_size >>> 
				(devSrc.GetData(), devDst.GetData(), width, height);
		}
		hipDeviceSynchronize();
		//watch.stop();
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		CUDA_CHECK_ERROR;

		devDst.CopyToHost(imgDst.data, imgDst.width, imgDst.width, imgDst.height);

		hipEventElapsedTime(&inc, start, stop);
		//inc = watch.getAverageTime();
		inc /= (float)nRepeatCount;
		printf("%dx%d , %dx%d , proc_count=%d, cache=%d, BLOCK_SIZE=%d, %f ms , %f fps\n", width, height, FILTER_WIDTH, FILTER_HEIGHT, PROCESS_DATA_COUNT, PROCESS_DATA_COUNT + FILTER_HEIGHT - 1, BLOCK_SIZE, inc, 1000.0 / inc);
		sprintf(szPath, "../data/Lena_proc_%dx%d.raw", width, height);
		//imgDst.SaveRaw(szPath);

		sprintf(szPath, "../data/Lena_proc_%dx%d.txt", width, height);
		//imgDst.SaveText(szPath);

		DataT<DataType> imgVerify;
		imgVerify.MallocBuffer(width, height);
		Convolution(img.data, imgVerify.data, width, height, filter[0], FILTER_WIDTH, FILTER_HEIGHT);
		sprintf(szPath, "../data/Lena_proc_verify_%dx%d.txt", width, height);
		//imgVerify.SaveText(szPath);

		double dif = 0;
		for (int i = 0; i < img.width*img.height; i++) {
			int x = i % img.width;
			int y = i / img.width;
			if (x > FILTER_WIDTH/2 && x < width - FILTER_WIDTH/2 && y > FILTER_HEIGHT/2 && y < height - FILTER_HEIGHT/2)
				dif += abs(imgVerify.data[i] - imgDst.data[i]);
		}
		printf("verify dif =%f, avg-dif=%e\n", dif, dif/img.width/img.height);
		sprintf(szPath, "../data/Lena_proc_verify_%dx%d.txt", width, height);
		//imgVerify.SaveText(szPath);
		sprintf(szPath, "../data/Lena_proc_verify(%dx%d)_%dx%d.raw", FILTER_WIDTH, FILTER_HEIGHT, width, height);
		//imgVerify.SaveRaw(szPath);
#if 0
		FILE* fp = fopen("log.conv2D.csv", "at");
		if (fp) {
			fprintf(fp, "%dx%d, %d_%d, %d, %dx%d, %f\n", width, height, PROCESS_DATA_COUNT, PROCESS_DATA_COUNT + FILTER_HEIGHT - 1, BLOCK_SIZE, FILTER_WIDTH, FILTER_HEIGHT, inc);
			fclose(fp);
		}
		return inc;
#endif
	}
};

int stencil_121pt_v2(int argc, char** argv) {
	DISPLAY_FUNCTION("");
	printf("datatype=double\n");
	int size = 8192; if (argc > 1) size = atoi(argv[1]);
	const int P = 4;
	const int B = 128;
	stencil2d_121pt_v2::Test<double, P, B>(size, size);
	return 0;
}


